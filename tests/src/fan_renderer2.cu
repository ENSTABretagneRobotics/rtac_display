#include "hip/hip_runtime.h"
#include <rtac_base/types/SonarPing.h>
using namespace rtac;

#include <rtac_base/cuda/CudaVector.h>
using namespace rtac::cuda;

#include <rtac_display/Display.h>
#include <rtac_display/renderers/FanRenderer.h>
using namespace rtac::display;

template <typename T>
__global__ void fill_ping(PingView2D<T> ping, unsigned int oversampling = 1)
{
    ping(blockIdx.x, threadIdx.x) = ((blockIdx.x/oversampling + threadIdx.x/oversampling) & 0x1);
}

int main()
{
    //unsigned int N = 4;
    //unsigned int oversampling = 1;
    //unsigned int N = 16;
    //unsigned int oversampling = 4;
    unsigned int N = 32;
    unsigned int oversampling = 4;
    Ping2D<float,CudaVector> p0(Linspace<float>(0.0f, 10.0f, N),
                               CudaVector<float>::linspace(-0.25*3.14, 0.25*3.14, N));
    fill_ping<<<p0.height(), p0.width()>>>(p0.view(), oversampling);
    hipDeviceSynchronize();

    Display display;
    auto renderer = display.create_renderer<FanRenderer>(View::Create());
    renderer->set_ping(p0);
    //renderer->disable_bearing_map();

    while(!display.should_close()) {
        display.draw();
    }

    return 0;
}


