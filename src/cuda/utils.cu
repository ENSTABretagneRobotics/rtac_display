#include <rtac_display/cuda/utils.h>

#include <hip/hip_runtime.h> // check if needed
#include <cuda_gl_interop.h>

namespace rtac { namespace display { namespace cuda {

void copy_to_gl(GLuint bufferId, const void* cudaDevicePtr, size_t byteCount)
{
    hipGraphicsResource* cudaResource(NULL);
    void* GLdevicePtr = NULL;
    size_t accessibleSize = 0;

    rtac::cuda::check_error( hipGraphicsGLRegisterBuffer(
        &cudaResource, bufferId, hipGraphicsRegisterFlagsWriteDiscard));
    rtac::cuda::check_error( hipGraphicsMapResources(1, &cudaResource));
    
    rtac::cuda::check_error( hipGraphicsResourceGetMappedPointer(
        &GLdevicePtr, &accessibleSize, cudaResource));
    
    if(accessibleSize < byteCount) {
        std::ostringstream oss;
        oss << "GL buffer not big enough for copy (needs " << byteCount 
            << "b, buffer is " << accessibleSize << "b)";
        throw std::runtime_error(oss.str());
    }

    rtac::cuda::check_error(hipMemcpy(GLdevicePtr, cudaDevicePtr, byteCount,
                              hipMemcpyDeviceToDevice));

    rtac::cuda::check_error( hipGraphicsUnmapResources(1, &cudaResource));
    rtac::cuda::check_error( hipGraphicsUnregisterResource(cudaResource));
}

void copy_from_gl(void* cudaDevicePtr, GLuint bufferId, size_t byteCount)
{
    hipGraphicsResource* cudaResource(NULL);
    void* GLdevicePtr = NULL;
    size_t accessibleSize = 0;

    rtac::cuda::check_error( hipGraphicsGLRegisterBuffer(
        &cudaResource, bufferId, hipGraphicsRegisterFlagsWriteDiscard));
    rtac::cuda::check_error( hipGraphicsMapResources(1, &cudaResource));
    
    rtac::cuda::check_error( hipGraphicsResourceGetMappedPointer(
        &GLdevicePtr, &accessibleSize, cudaResource));
    
    if(accessibleSize < byteCount) {
        std::ostringstream oss;
        oss << "GL buffer not big enough for copy (needs " << byteCount 
            << "b, buffer is " << accessibleSize << "b)";
        throw std::runtime_error(oss.str());
    }

    rtac::cuda::check_error(hipMemcpy(cudaDevicePtr, GLdevicePtr, byteCount,
                              hipMemcpyDeviceToDevice));

    rtac::cuda::check_error( hipGraphicsUnmapResources(1, &cudaResource));
    rtac::cuda::check_error( hipGraphicsUnregisterResource(cudaResource));
}

}; //namespace cuda
}; //namespace display
}; //namespace rtac
