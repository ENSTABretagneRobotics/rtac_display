#include <rtac_display/cuda/utils.h>

#include <hip/hip_runtime.h> // check if needed
#include <cuda_gl_interop.h>

namespace rtac { namespace display { namespace cuda {

void copy_to_gl(GLuint bufferId, const void* cudaDevicePtr, size_t byteCount)
{
    hipGraphicsResource* cudaResource(NULL);
    void* GLdevicePtr = NULL;
    size_t accessibleSize = 0;

    CUDA_CHECK( hipGraphicsGLRegisterBuffer(
        &cudaResource, bufferId, hipGraphicsRegisterFlagsWriteDiscard));
    CUDA_CHECK( hipGraphicsMapResources(1, &cudaResource));
    
    CUDA_CHECK( hipGraphicsResourceGetMappedPointer(
        &GLdevicePtr, &accessibleSize, cudaResource));
    
    if(accessibleSize < byteCount) {
        std::ostringstream oss;
        oss << "GL buffer not big enough for copy (needs " << byteCount 
            << "b, buffer is " << accessibleSize << "b)";
        throw std::runtime_error(oss.str());
    }

    CUDA_CHECK(hipMemcpy(GLdevicePtr, cudaDevicePtr, byteCount,
                              hipMemcpyDeviceToDevice));

    CUDA_CHECK( hipGraphicsUnmapResources(1, &cudaResource));
    CUDA_CHECK( hipGraphicsUnregisterResource(cudaResource));
}

void copy_from_gl(void* cudaDevicePtr, GLuint bufferId, size_t byteCount)
{
    hipGraphicsResource* cudaResource(NULL);
    void* GLdevicePtr = NULL;
    size_t accessibleSize = 0;

    CUDA_CHECK( hipGraphicsGLRegisterBuffer(
        &cudaResource, bufferId, hipGraphicsRegisterFlagsWriteDiscard));
    CUDA_CHECK( hipGraphicsMapResources(1, &cudaResource));
    
    CUDA_CHECK( hipGraphicsResourceGetMappedPointer(
        &GLdevicePtr, &accessibleSize, cudaResource));
    
    if(accessibleSize < byteCount) {
        std::ostringstream oss;
        oss << "GL buffer not big enough for copy (needs " << byteCount 
            << "b, buffer is " << accessibleSize << "b)";
        throw std::runtime_error(oss.str());
    }

    CUDA_CHECK(hipMemcpy(cudaDevicePtr, GLdevicePtr, byteCount,
                              hipMemcpyDeviceToDevice));

    CUDA_CHECK( hipGraphicsUnmapResources(1, &cudaResource));
    CUDA_CHECK( hipGraphicsUnregisterResource(cudaResource));
}

}; //namespace cuda
}; //namespace display
}; //namespace rtac
